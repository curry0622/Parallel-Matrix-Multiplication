
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <time.h>

void multiply_cache_friendly(int *a, int *b, int *c, int m, int n, int l) {
    for (int i = 0; i < m; i++) {
        for (int k = 0; k < n; k++) {
            int r = a[i * n + k];
            for (int j = 0; j < l; j++)
                c[i * l + j] += r * b[k * l + j];
        }
    }
}

int main(int argc, char *argv[]) {
    // Timer start
    clock_t prog_t, cpu_t;
    prog_t = clock();

    // Read inputs
    assert(argc == 3);
    FILE *f = fopen(argv[1], "r");
    assert(f);
    int m, n, l;
    fscanf(f, "%d %d %d", &m, &n, &l);
    int *a = new int[m * n];
    int *b = new int[n * l];
    int *c = new int[m * l];
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++)
            fscanf(f, "%d", &a[i * n + j]);
    for (int i = 0; i < n; i++)
        for (int j = 0; j < l; j++)
            fscanf(f, "%d", &b[i * l + j]);
    fclose(f);

    // multiply a and b
    cpu_t = clock();
    multiply_cache_friendly(a, b, c, m, n, l);
    cpu_t = clock() - cpu_t;
    
    // Output c to file
    f = fopen(argv[2], "w");
    assert(f);
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < l; j++)
            fprintf(f, "%ld ", c[i * l + j]);
        fprintf(f, "\n");
    }
    fclose(f);

    // Timer end
    prog_t = clock() - prog_t;
    printf("Total time: %f seconds\n", (double)prog_t / CLOCKS_PER_SEC);
    printf("CPU time: %f seconds\n", (double)cpu_t / CLOCKS_PER_SEC);
    printf("IO time: %f seconds\n", (double)(prog_t - cpu_t) / CLOCKS_PER_SEC);

    return 0;
}
