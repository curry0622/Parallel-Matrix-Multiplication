
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <time.h>

#define B 32

int *a, *b, *c;
int m, n, l;

void input(char *filename) {
    // Open file
    FILE *f = fopen(filename, "r");
    assert(f);

    // Read m, n, l
    fscanf(f, "%d %d %d", &m, &n, &l);

    // Allocate memory
    hipHostMalloc((void**)&a, m * n * sizeof(int), hipHostMallocDefault);
    hipHostMalloc((void**)&b, n * l * sizeof(int), hipHostMallocDefault);
    hipHostMalloc((void**)&c, m * l * sizeof(int), hipHostMallocDefault);

    // Read a
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++)
            fscanf(f, "%d", &a[i * n + j]);

    // Read b
    for (int i = 0; i < n; i++)
        for (int j = 0; j < l; j++)
            fscanf(f, "%d", &b[i * l + j]);

    // Close file
    fclose(f);
}

void output(char *filename) {
    // Open file
    FILE *f = fopen(filename, "w");
    assert(f);

    // Write c
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < l; j++)
            fprintf(f, "%d ", c[i * l + j]);
        fprintf(f, "\n");
    }

    // Close file
    fclose(f);
}

void dump_time(double io_t, double mul_t, double total_t) {
    FILE *f = fopen("time.txt", "w");
    assert(f);
    fprintf(f, "%f\n", io_t);
    fprintf(f, "%f\n", mul_t);
    fprintf(f, "%f\n", total_t);
    fclose(f);
}

__global__ void multiply_naive(int *d_a, int *d_b, int *d_c, int m, int n, int l) {
    int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
    int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(row_idx >= m || col_idx >= l) return;
    int sum = 0;
    for (int i = 0; i < n; i++)
        sum += d_a[row_idx * n + i] * d_b[i * l + col_idx];
    d_c[row_idx * l + col_idx] = sum;
}

int main(int argc, char *argv[]) {
    // Timer start
    clock_t total_t, input_t, output_t;
    total_t = clock();

    // Argument check
    assert(argc == 3);

    // Read inputs
    input_t = clock();
    input(argv[1]);
    input_t = clock() - input_t;

    // Allocate memory on device
    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, m * n * sizeof(int));
    hipMalloc((void**)&d_b, n * l * sizeof(int));
    hipMalloc((void**)&d_c, m * l * sizeof(int));

    // Copy inputs to device
    hipMemcpy(d_a, a, m * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * l * sizeof(int), hipMemcpyHostToDevice);

    // Multiply a and b
    printf("Multiplying a and b on GPU...\n");
    multiply_naive<<<dim3(l / B + 1, m / B + 1), dim3(B, B)>>>(d_a, d_b, d_c, m, n, l);
    printf("done\n");

    // Copy result from device to host
    hipMemcpy(c, d_c, m * l * sizeof(int), hipMemcpyDeviceToHost);
    
    // Output c to file
    output_t = clock();
    output(argv[2]);
    output_t = clock() - output_t;

    // Print time
    total_t = clock() - total_t;
    printf("Time: %f\n", (double)total_t / CLOCKS_PER_SEC);
    printf("IO time: %f\n", (double)(input_t + output_t) / CLOCKS_PER_SEC);
    printf("GPU time: %f\n", (double)(total_t - input_t - output_t) / CLOCKS_PER_SEC);
    dump_time(
        (double)input_t / CLOCKS_PER_SEC,
        (double)(total_t - input_t - output_t) / CLOCKS_PER_SEC,
        (double)total_t / CLOCKS_PER_SEC
    );

    return 0;
}
