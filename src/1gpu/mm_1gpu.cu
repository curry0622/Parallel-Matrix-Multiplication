
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define B 32

int *a, *b, *c;
int m, n, l;

void input(char *filename) {
    // Open file
    FILE *f = fopen(filename, "r");
    assert(f);

    // Read m, n, l
    fscanf(f, "%d %d %d", &m, &n, &l);

    // Allocate memory
    hipHostMalloc((void**)&a, m * n * sizeof(int), hipHostMallocDefault);
    hipHostMalloc((void**)&b, n * l * sizeof(int), hipHostMallocDefault);
    hipHostMalloc((void**)&c, m * l * sizeof(int), hipHostMallocDefault);

    // Read a
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++)
            fscanf(f, "%d", &a[i * n + j]);

    // Read b
    for (int i = 0; i < n; i++)
        for (int j = 0; j < l; j++)
            fscanf(f, "%d", &b[i * l + j]);

    // Close file
    fclose(f);
}

void output(char *filename) {
    // Open file
    FILE *f = fopen(filename, "w");
    assert(f);

    // Write c
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < l; j++)
            fprintf(f, "%ld ", c[i * l + j]);
        fprintf(f, "\n");
    }

    // Close file
    fclose(f);
}

__global__ void multiply_gpu(int *d_a, int *d_b, int *d_c, int m, int n, int l) {
    int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
    int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(row_idx >= m || col_idx >= l) return;
    int sum = 0;
    for (int i = 0; i < n; i++)
        sum += d_a[row_idx * n + i] * d_b[i * l + col_idx];
    d_c[row_idx * l + col_idx] = sum;
}

void multiply_naive(int *a, int *b, int *c, int m, int n, int l) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < l; j++) {
            int sum = 0;
            for (int k = 0; k < n; k++)
                sum += a[i * n + k] * b[k * l + j];
            c[i * l + j] = sum;
        }
    }
}

void multiply_cache_friendly(int *a, int *b, int *c, int m, int n, int l) {
    for (int i = 0; i < m; i++) {
        for (int k = 0; k < n; k++) {
            int r = a[i * n + k];
            for (int j = 0; j < l; j++)
                c[i * l + j] += r * b[k * l + j];
        }
    }
}

int main(int argc, char *argv[]) {
    // Argument check
    assert(argc == 3);

    // Read inputs
    input(argv[1]);

    // Allocate memory on device
    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, m * n * sizeof(int));
    hipMalloc((void**)&d_b, n * l * sizeof(int));
    hipMalloc((void**)&d_c, m * l * sizeof(int));

    // Multiply a and b
    printf("Multiplying a and b on GPU...");
    multiply_gpu<<<dim3(l / B, m / B), dim3(B, B)>>>(d_a, d_b, d_c, m, n, l);
    printf("done\n");

    // Copy result from device to host
    hipMemcpy(c, d_c, m * l * sizeof(int), hipMemcpyDeviceToHost);
    
    // Output c to file
    output(argv[2]);

    return 0;
}
