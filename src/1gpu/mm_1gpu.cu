
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <time.h>

#define B 32

int *a, *b, *c;
int m, n, l;

void input(char *filename) {
    // Open file
    FILE *f = fopen(filename, "r");
    assert(f);

    // Read m, n, l
    fscanf(f, "%d %d %d", &m, &n, &l);

    // Allocate memory
    hipHostMalloc((void**)&a, m * n * sizeof(int), hipHostMallocDefault);
    hipHostMalloc((void**)&b, n * l * sizeof(int), hipHostMallocDefault);
    hipHostMalloc((void**)&c, m * l * sizeof(int), hipHostMallocDefault);

    // Read a
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++)
            fscanf(f, "%d", &a[i * n + j]);

    // Read b
    for (int i = 0; i < n; i++)
        for (int j = 0; j < l; j++)
            fscanf(f, "%d", &b[i * l + j]);

    // Close file
    fclose(f);
}

void output(char *filename) {
    // Open file
    FILE *f = fopen(filename, "w");
    assert(f);

    // Write c
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < l; j++)
            fprintf(f, "%d ", c[i * l + j]);
        fprintf(f, "\n");
    }

    // Close file
    fclose(f);
}

__global__ void multiply_naive(int *d_a, int *d_b, int *d_c, int m, int n, int l) {
    int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
    int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(row_idx >= m || col_idx >= l) return;
    int sum = 0;
    for (int i = 0; i < n; i++)
        sum += d_a[row_idx * n + i] * d_b[i * l + col_idx];
    d_c[row_idx * l + col_idx] = sum;
}

int main(int argc, char *argv[]) {
    // Timer start
    clock_t prog_t, input_t, output_t;
    prog_t = clock();

    // Argument check
    assert(argc == 3);

    // Read inputs
    input_t = clock();
    input(argv[1]);
    input_t = clock() - input_t;

    // Allocate memory on device
    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, m * n * sizeof(int));
    hipMalloc((void**)&d_b, n * l * sizeof(int));
    hipMalloc((void**)&d_c, m * l * sizeof(int));

    // Copy inputs to device
    hipMemcpy(d_a, a, m * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * l * sizeof(int), hipMemcpyHostToDevice);

    // Multiply a and b
    printf("Multiplying a and b on GPU...\n");
    multiply_naive<<<dim3(l / B + 1, m / B + 1), dim3(B, B)>>>(d_a, d_b, d_c, m, n, l);
    printf("done\n");

    // Copy result from device to host
    hipMemcpy(c, d_c, m * l * sizeof(int), hipMemcpyDeviceToHost);
    
    // Output c to file
    output_t = clock();
    output(argv[2]);
    output_t = clock() - output_t;

    // Print time
    prog_t = clock() - prog_t;
    printf("Time: %f\n", (double)prog_t / CLOCKS_PER_SEC);
    printf("IO time: %f\n", (double)(input_t + output_t) / CLOCKS_PER_SEC);
    printf("GPU time: %f\n", (double)(prog_t - input_t - output_t) / CLOCKS_PER_SEC);

    return 0;
}
